#include "hip/hip_runtime.h"
//
// Created by BabdCatha on 19/10/2021.
//

#include <iostream>
#include "Root.cuh"

Root::Root(int coordX, int coordY, const Scale &scale, sf::RenderWindow * window, sf::Color color) : value(XYtoComplex(coordX, coordY)), coordX(coordX), coordY(coordY), scale(scale){
	this->window = window;
	sprite = sf::CircleShape(ROOT_RADIUS);
	rootColor = color;
	sprite.setPosition((float)coordX, (float)coordY);
	sprite.setFillColor(sf::Color::Yellow);
	selected = false;
}

Root::Root(hipFloatComplex z, const Scale &scale, sf::RenderWindow * window, sf::Color color) : value(z), scale(scale){
	int * res = complextoXY(z, this->scale);
	coordX = res[0];
	coordY = res[1];
	this->window = window;
	sprite = sf::CircleShape(ROOT_RADIUS);
	rootColor = color;
	sprite.setPosition((float)coordX, (float)coordY);
	sprite.setFillColor(sf::Color::Yellow);
	selected = false;
}

Root::Root() : value(make_hipFloatComplex(0, 0)), scale(make_hipFloatComplex(0, 0), 1, 1, 1920, 1080){
	coordX = 0;
	coordY = 0;
	this->window = nullptr;
	sprite = sf::CircleShape(ROOT_RADIUS);
	sprite.setPosition((float)coordX, (float)coordY);
	sprite.setFillColor(sf::Color::Blue);
	rootColor = sf::Color::Yellow;
	selected = false;
}

void Root::draw() {
	window->draw(sprite);
}

bool Root::overlaps(int e_x, int e_y) const {
	if(abs(e_x - ROOT_RADIUS - coordX) < ROOT_RADIUS && abs(e_y - ROOT_RADIUS - coordY) < ROOT_RADIUS){
		return true;
	}
	return false;
}

void Root::updatePosition(int n_x, int n_y) {
	coordX = n_x - ROOT_RADIUS;
	coordY = n_y - ROOT_RADIUS;
	value = XYtoComplex(coordX, coordY);
	sprite.setPosition((float)coordX, (float)coordY);
}

bool Root::isSelected() const {
	return selected;
}

void Root::setSelected(bool n_selected) {
	Root::selected = n_selected;
}

hipFloatComplex Root::getValue() {
	return value;
}

hipFloatComplex Root::XYtoComplex(int x, int y) {
	hipFloatComplex res = scale.getCenter();
	//res += (x - (scale.getScreenWidth() / 2)) * (2 * (double)scale.getWidth() / (double)scale.getScreenWidth());
	//res += 1i * ((scale.getScreenHeight() / 2) - y) * (2 * (double)scale.getHeight() / (double)scale.getScreenHeight());

	res = hipCaddf(res, make_hipFloatComplex((x - (scale.getScreenWidth() / 2)) * (2 * (float)scale.getWidth() / (float)scale.getScreenWidth()), 0));
	res = hipCaddf(res, make_hipFloatComplex(0, ((scale.getScreenHeight() / 2) - y) * (2 * (float)scale.getHeight() / (float)scale.getScreenHeight())));

	return res;
}

sf::Color Root::getRootColor() {
	return rootColor;
}

int * Root::complextoXY(hipFloatComplex z, Scale &scaleParameter){
	int * res = (int*)malloc(2*sizeof(int));

	res[0] = static_cast<int>(hipCrealf(z) / scaleParameter.getWidth() * scaleParameter.getScreenWidth() + scaleParameter.getScreenWidth() / 2);
	res[1] = static_cast<int>(hipCimagf(z) / scaleParameter.getHeight() * scaleParameter.getScreenHeight() + scaleParameter.getScreenHeight() / 2);

	return res;
}