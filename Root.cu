#include "hip/hip_runtime.h"
//
// Created by BabdCatha on 19/10/2021.
//

#include <iostream>
#include "Root.cuh"

Root::Root(int coordX, int coordY, const Scale &scale, sf::RenderWindow * window, sf::Color color) : value(XYtoComplex(coordX, coordY)), coordX(coordX), coordY(coordY), scale(scale){
	this->window = window;
	sprite = sf::CircleShape(ROOT_RADIUS);
	rootColor = color;
	sprite.setPosition((float)coordX, (float)coordY);
	sprite.setFillColor(sf::Color::Blue);
	selected = false;
}

Root::Root(hipDoubleComplex z, const Scale &scale, sf::RenderWindow * window, sf::Color color) : value(z), scale(scale){
	int * res = complextoXY(z, this->scale);
	coordX = res[0];
	coordY = res[1];
	this->window = window;
	sprite = sf::CircleShape(ROOT_RADIUS);
	rootColor = color;
	sprite.setPosition((float)coordX, (float)coordY);
	sprite.setFillColor(sf::Color::Blue);
	selected = false;
}

Root::Root() : value(make_hipDoubleComplex(0, 0)), scale(make_hipDoubleComplex(0, 0), 1, 1, 1920, 1080){
	coordX = 0;
	coordY = 0;
	this->window = nullptr;
	sprite = sf::CircleShape(ROOT_RADIUS);
	sprite.setPosition((float)coordX, (float)coordY);
	sprite.setFillColor(sf::Color::Blue);
	rootColor = sf::Color::Blue;
	selected = false;
}

void Root::draw() {
	window->draw(sprite);
}

bool Root::overlaps(int e_x, int e_y) const {
	if(abs(e_x - ROOT_RADIUS - coordX) < ROOT_RADIUS && abs(e_y - ROOT_RADIUS - coordY) < ROOT_RADIUS){
		return true;
	}
	return false;
}

void Root::updatePosition(int n_x, int n_y) {
	coordX = n_x - ROOT_RADIUS;
	coordY = n_y - ROOT_RADIUS;
	value = XYtoComplex(coordX, coordY);
	sprite.setPosition((float)coordX, (float)coordY);
}

bool Root::isSelected() const {
	return selected;
}

void Root::setSelected(bool n_selected) {
	Root::selected = n_selected;
}

hipDoubleComplex Root::getValue() {
	return value;
}

hipDoubleComplex Root::XYtoComplex(int x, int y) {
	hipDoubleComplex res = scale.getCenter();
	//res += (x - (scale.getScreenWidth() / 2)) * (2 * (double)scale.getWidth() / (double)scale.getScreenWidth());
	//res += 1i * ((scale.getScreenHeight() / 2) - y) * (2 * (double)scale.getHeight() / (double)scale.getScreenHeight());

	res = hipCadd(res, make_hipDoubleComplex((x - (scale.getScreenWidth() / 2)) * (2 * (double)scale.getWidth() / (double)scale.getScreenWidth()), 0));
	res = hipCadd(res, make_hipDoubleComplex(0, ((scale.getScreenHeight() / 2) - y) * (2 * (double)scale.getHeight() / (double)scale.getScreenHeight())));

	return res;
}

sf::Color Root::getRootColor() {
	return rootColor;
}

int * Root::complextoXY(hipDoubleComplex z, Scale &scaleParameter){
	int * res = (int*)malloc(2*sizeof(int));

	res[0] = static_cast<int>(hipCreal(z) / scaleParameter.getWidth() * scaleParameter.getScreenWidth() + scaleParameter.getScreenWidth() / 2);
	res[1] = static_cast<int>(hipCimag(z) / scaleParameter.getHeight() * scaleParameter.getScreenHeight() + scaleParameter.getScreenHeight() / 2);

	return res;
}