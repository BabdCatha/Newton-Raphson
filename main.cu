#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <SFML/Window.hpp>
#include <hip/hip_complex.h>

#include "Root.cuh"
#include "Scale.cuh"
#include "Polynomial.cuh"

#define SCREEN_WIDTH 1920
#define SCREEN_HEIGHT 1080
#define MAX_FRAMERATE 60

#define NUMBER_OF_ITERATIONS 15
//#define BASE_DEGREE 3

hipFloatComplex XYtoComplex(int x, int y, Scale &scale);
void initScreen(Scale &scale, hipFloatComplex **complexScreen);
void getColorMap(Polynomial *P, hipFloatComplex **complexScreen, sf::Image *image);
__global__ void performNewtonStep(Polynomial *P, hipFloatComplex **complexScreen);

int main(){

	//Creating the complex screen, and making it accessible to the device
	hipFloatComplex ** complexScreen;
	hipMallocManaged(&complexScreen, SCREEN_WIDTH*sizeof(hipFloatComplex&));
	for(int i = 0; i < SCREEN_WIDTH; i++){
		hipMallocManaged(&(complexScreen[i]), SCREEN_HEIGHT*sizeof(hipFloatComplex));
	}

	//Creating the main window, in fullscreen WUXGA- mode
	sf::RenderWindow window(sf::VideoMode(SCREEN_WIDTH, SCREEN_HEIGHT), "Newton-Raphson fractal", sf::Style::Fullscreen);
	window.setFramerateLimit(MAX_FRAMERATE); //60Hz max

	//Creating the background image
	//It is stored in RAM, and this is where we will update each pixel's color
	sf::Image backgroundImage;
	backgroundImage.create(SCREEN_WIDTH, SCREEN_HEIGHT, sf::Color(255,0,0));

	//Creating the background texture
	//This texture is a VRAM copy of the image, and is applied to a 1920x1080 sprite, to be drawn as the background
	sf::Texture backgroundTexture;
	backgroundTexture.loadFromImage(backgroundImage);

	//Creating the background sprite
	sf::Sprite backgroundSprite;
	backgroundSprite.setTexture(backgroundTexture);

	//Creating the current scale
	Scale currentScale(make_hipFloatComplex(0, 0), 1*5, 1*2.8125, SCREEN_WIDTH, SCREEN_HEIGHT);

	//Test variables
	Root z1(make_hipFloatComplex(1, 0), currentScale, &window, sf::Color(0, 0, 255));
	Root z2(make_hipFloatComplex(-0.5, 0.86602540378443), currentScale, &window, sf::Color(255, 0, 0));
	Root z3(make_hipFloatComplex(-0.5, -0.86602540378443), currentScale, &window, sf::Color(0, 255, 0));
	Root z4(100, 400, currentScale, &window, sf::Color(61, 15, 97));
	Root z5(100, 500, currentScale, &window, sf::Color(97, 15, 72));

	Root liste[] = {z1, z2, z3};

	Polynomial *P;
	hipMallocManaged(&P, sizeof(Polynomial));
	*P = Polynomial(3, liste);

	//Calculating variables for the number of blocks and threads
	int blockSize = 256;
	int numBlocks = (SCREEN_WIDTH*SCREEN_HEIGHT + blockSize - 1) / blockSize;

	while(window.isOpen()){
		sf::Event event{};
		while(window.pollEvent(event)){
			//If the user quits the app, we close it
			if(event.type == sf::Event::Closed) {
				window.close();
			}

			//If the user performs a left click, we set up the corresponding flag in the selected root, if any
			if(event.type == sf::Event::MouseButtonPressed && event.mouseButton.button == sf::Mouse::Left){
				P->leftMouseButtonPressed(event);
			}
			//If the user releases the left mouse button, we clear the corresponding flag
			if(event.type == sf::Event::MouseButtonReleased && event.mouseButton.button == sf::Mouse::Left){
				P->leftMouseButtonReleased();
			}

			//If the mouse moves while the left mouse button is pressed, we make each root handle the movement
			if(event.type == sf::Event::MouseMoved && P->getIsLeftMouseButtonPressed()){
				P->update(event);
			}

			//If we want to re-draw the background, on a right click
			if(event.type == sf::Event::MouseButtonPressed && event.mouseButton.button == sf::Mouse::Right){
				initScreen(currentScale, complexScreen);

				for(int i = 0; i < SCREEN_WIDTH; i++){
					hipMemPrefetchAsync(complexScreen[i], SCREEN_HEIGHT*sizeof(hipFloatComplex), 0);
				}
				hipMemPrefetchAsync(complexScreen, SCREEN_WIDTH*sizeof(hipFloatComplex*), 0);

				hipDeviceSynchronize();

				for(int i = 0; i < NUMBER_OF_ITERATIONS; i++){

					performNewtonStep<<<numBlocks,blockSize>>>(P, complexScreen);
					hipDeviceSynchronize();

				}

				for(int i = 0; i < SCREEN_WIDTH; i++){
					hipMemPrefetchAsync(complexScreen[i], SCREEN_HEIGHT*sizeof(hipFloatComplex), hipCpuDeviceId);
				}
				hipMemPrefetchAsync(complexScreen, SCREEN_WIDTH*sizeof(hipFloatComplex*), hipCpuDeviceId);

				hipDeviceSynchronize();

				getColorMap(P, complexScreen, &backgroundImage);
				backgroundTexture.update(backgroundImage);

			}
		}

		//Cleaning the window of any previous renderings
		window.clear();

		//Drawing the background image
		window.draw(backgroundSprite);

		//This is where the roots should be drawn
		P->drawRoots();

		//Once all the modifications have been done, the window is drawn again
		window.display();

	}

	return 0;
}

hipFloatComplex XYtoComplex(int x, int y, Scale &scale) {

	hipFloatComplex res = scale.getCenter();

	res = hipCaddf(res, make_hipFloatComplex((x - (scale.getScreenWidth() / 2)) * (2 * (double)scale.getWidth() / (double)scale.getScreenWidth()), 0));
	res = hipCaddf(res, make_hipFloatComplex(0, ((scale.getScreenHeight() / 2) - y) * (2 * (double)scale.getHeight() / (double)scale.getScreenHeight())));

	return res;
}

void initScreen(Scale &scale, hipFloatComplex **complexScreen){
	for (int i = 0; i < SCREEN_WIDTH; i++)
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			complexScreen[i][j] = XYtoComplex(i, j, scale);
		}
}

void getColorMap(Polynomial *P, hipFloatComplex **complexScreen, sf::Image *image){
	for (int i = 0; i < SCREEN_WIDTH; i++)
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			image->setPixel(i, j, P->findClosestRootColor(complexScreen[i][j]));
		}
}

__global__
void performNewtonStep(Polynomial *P, hipFloatComplex **complexScreen){

	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for(unsigned int i = index; i < SCREEN_WIDTH*SCREEN_HEIGHT; i += stride){

		unsigned int x = i%SCREEN_WIDTH;
		unsigned int y = i/SCREEN_WIDTH;

		hipFloatComplex alpha = complexScreen[x][y];

		hipFloatComplex val = make_hipFloatComplex(0, 0);
		P->evaluate(alpha, &val);

		hipFloatComplex valD = make_hipFloatComplex(0, 0);
		P->evaluate_derivative(alpha, &valD);

		complexScreen[x][y] = hipCsubf(complexScreen[x][y], hipCdivf(val, valD));
	}

}