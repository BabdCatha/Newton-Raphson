#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <SFML/Window.hpp>
#include <hip/hip_complex.h>
#include <c++/10/iostream>

#include "Root.cuh"
#include "Scale.cuh"
#include "Polynomial.cuh"

#define SCREEN_WIDTH 1920
#define SCREEN_HEIGHT 1080
#define MAX_FRAMERATE 60

#define NUMBER_OF_ITERATIONS 20
//#define BASE_DEGREE 3

//sf::Color screen[SCREEN_WIDTH][SCREEN_HEIGHT];

hipDoubleComplex XYtoComplex(int x, int y, Scale &scale);
void initScreen(Scale &scale, hipDoubleComplex **complexScreen);
void getColorMap(Polynomial *P, hipDoubleComplex **complexScreen, sf::Image *image);
__global__ void performNewtonStep(Polynomial *P, hipDoubleComplex **complexScreen);

int main(){

	//Creating the complex screen, and making it accessible to the device
	hipDoubleComplex ** complexScreen;
	//hipMallocManaged(&complexScreen, SCREEN_WIDTH*SCREEN_HEIGHT*sizeof(std::complex<double>));
	hipMallocManaged(&complexScreen, SCREEN_WIDTH*sizeof(hipDoubleComplex&));
	for(int i = 0; i < SCREEN_WIDTH; i++){
		hipMallocManaged(&(complexScreen[i]), SCREEN_HEIGHT*sizeof(hipDoubleComplex));
	}

	//Creating the main window, in fullscreen WUXGA- mode
	sf::RenderWindow window(sf::VideoMode(SCREEN_WIDTH, SCREEN_HEIGHT), "Newton-Raphson fractal", sf::Style::Fullscreen);
	window.setFramerateLimit(MAX_FRAMERATE); //60Hz max

	//Creating the background image
	//It is stored in RAM, and this is where we will update each pixel's color
	sf::Image backgroundImage;
	backgroundImage.create(SCREEN_WIDTH, SCREEN_HEIGHT, sf::Color(255,0,0));

	//Creating the background texture
	//This texture is a VRAM copy of the image, and is applied to a 1920x1080 sprite, to be drawn as the background
	sf::Texture backgroundTexture;
	backgroundTexture.loadFromImage(backgroundImage);

	//Creating the background sprite
	sf::Sprite backgroundSprite;
	backgroundSprite.setTexture(backgroundTexture);

	//Creating the current scale
	Scale currentScale(make_hipDoubleComplex(0, 0), 1*5, 1*2.8125, SCREEN_WIDTH, SCREEN_HEIGHT);

	//Test variables
	Root z1(make_hipDoubleComplex(1, 0), currentScale, &window, sf::Color(0, 0, 255));
	Root z2(make_hipDoubleComplex(-0.5, 0.86602540378443), currentScale, &window, sf::Color(255, 0, 0));
	Root z3(make_hipDoubleComplex(-0.5, -0.86602540378443), currentScale, &window, sf::Color(0, 255, 0));
	Root z4(100, 400, currentScale, &window, sf::Color(61, 15, 97));
	Root z5(100, 500, currentScale, &window, sf::Color(97, 15, 72));

	Root liste[] = {z1, z2, z3};

	Polynomial *P;
	hipMallocManaged(&P, sizeof(Polynomial));
	*P = Polynomial(3, liste);

	while(window.isOpen()){
		sf::Event event{};
		while(window.pollEvent(event)){
			//If the user quits the app, we close it
			if(event.type == sf::Event::Closed) {
				window.close();
			}

			//If the user performs a left click, we set up the corresponding flag in the selected root, if any
			if(event.type == sf::Event::MouseButtonPressed && event.mouseButton.button == sf::Mouse::Left){
				P->leftMouseButtonPressed(event);
			}
			//If the user releases the left mouse button, we clear the corresponding flag
			if(event.type == sf::Event::MouseButtonReleased && event.mouseButton.button == sf::Mouse::Left){
				P->leftMouseButtonReleased();
			}

			//If the mouse moves while the left mouse button is pressed, we make each root handle the movement
			if(event.type == sf::Event::MouseMoved && P->getIsLeftMouseButtonPressed()){
				P->update(event);
			}

			//If we want to re-draw the background, on a right click
			if(event.type == sf::Event::MouseButtonPressed && event.mouseButton.button == sf::Mouse::Right){
				initScreen(currentScale, complexScreen);

				for(int i = 0; i < NUMBER_OF_ITERATIONS; i++){

					performNewtonStep<<<1,256>>>(P, complexScreen);
					hipDeviceSynchronize();

					hipError_t err = hipGetLastError();
					if (err != hipSuccess){
						std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
						return 0;
					}


					//std::cout << i << std::endl;

					getColorMap(P, complexScreen, &backgroundImage);
					backgroundTexture.update(backgroundImage);

					//Drawing the background image
					window.draw(backgroundSprite);

					//Once all the modifications have been done, the window is drawn again
					window.display();

				}
			}
		}

		//Cleaning the window of any previous renderings
		window.clear();

		//Drawing the background image
		window.draw(backgroundSprite);

		//This is where the roots should be drawn
		P->drawRoots();

		//Once all the modifications have been done, the window is drawn again
		window.display();

	}

	return 0;
}

hipDoubleComplex XYtoComplex(int x, int y, Scale &scale) {
	hipDoubleComplex res = scale.getCenter();

	//res += (x - (scale.getScreenWidth() / 2)) * (2 * (double)scale.getWidth() / (double)scale.getScreenWidth());
	//res += (1i * ((scale.getScreenHeight() / 2) - y) * (2 * (double)scale.getHeight() / (double)scale.getScreenHeight()));

	res = hipCadd(res, make_hipDoubleComplex((x - (scale.getScreenWidth() / 2)) * (2 * (double)scale.getWidth() / (double)scale.getScreenWidth()), 0));
	res = hipCadd(res, make_hipDoubleComplex(0, ((scale.getScreenHeight() / 2) - y) * (2 * (double)scale.getHeight() / (double)scale.getScreenHeight())));

	return res;
}

void initScreen(Scale &scale, hipDoubleComplex **complexScreen){
	for (int i = 0; i < SCREEN_WIDTH; i++)
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			complexScreen[i][j] = XYtoComplex(i, j, scale);
		}
}

void getColorMap(Polynomial *P, hipDoubleComplex **complexScreen, sf::Image *image){
	for (int i = 0; i < SCREEN_WIDTH; i++)
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			image->setPixel(i, j, P->findClosestRootColor(complexScreen[i][j]));
		}
}

__global__
void performNewtonStep(Polynomial *P, hipDoubleComplex **complexScreen){

	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for(unsigned int i = index; i < SCREEN_WIDTH*SCREEN_HEIGHT; i += stride){
		unsigned int x = i%SCREEN_WIDTH;
		unsigned int y = i/SCREEN_WIDTH;

		hipDoubleComplex alpha = complexScreen[x][y];

		hipDoubleComplex val = make_hipDoubleComplex(0, 0);
		P->evaluate(alpha, &val);  //CRASHES

		hipDoubleComplex valD = make_hipDoubleComplex(0, 0);
		P->evaluate_derivative(alpha, &valD); //CRASHES

		complexScreen[x][y] = hipCsub(complexScreen[x][y], hipCdiv(val, valD));  //CRASHES
	}

	/*for (int i = 0; i < SCREEN_WIDTH; i++)
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			hipDoubleComplex alpha = complexScreen[i][j];

			hipDoubleComplex val = make_hipDoubleComplex(0, 0);
			P->evaluate(alpha, &val);  //CRASHES

			hipDoubleComplex valD = make_hipDoubleComplex(0, 0);
			P->evaluate_derivative(alpha, &valD); //CRASHES

			complexScreen[i][j] = hipCsub(complexScreen[i][j], hipCdiv(val, valD));  //CRASHES

			//-----*complexScreen[i][j] -= (P->evaluate(alpha)/P->evaluate_derivative(alpha));
		}*/
}