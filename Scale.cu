//
// Created by BabdCatha on 19/10/2021.
//

#include "Scale.cuh"

Scale::Scale(hipFloatComplex cent, double w, double h, int screenWidth, int screenHeight) : center(cent) {
	width = w;
	height = h;
	this->screenWidth = screenWidth;
	this->screenHeight = screenHeight;
}

const hipFloatComplex &Scale::getCenter() const {
	return center;
}

double Scale::getWidth() const {
	return width;
}

double Scale::getHeight() const {
	return height;
}

double Scale::getScreenWidth() const {
	return screenWidth;
}

double Scale::getScreenHeight() const {
	return screenHeight;
}

