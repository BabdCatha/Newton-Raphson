//
// Created by BabdCatha on 19/10/2021.
//

#include "Scale.cuh"

Scale::Scale(hipFloatComplex cent, float w, float h, int screenWidth, int screenHeight) : center(cent) {
	width = w;
	height = h;
	this->screenWidth = screenWidth;
	this->screenHeight = screenHeight;
}

const hipFloatComplex &Scale::getCenter() const {
	return center;
}

__device__
const hipFloatComplex &Scale::getCenterD() const {
	return center;
}

float Scale::getWidth() const {
	return width;
}

float Scale::getHeight() const {
	return height;
}

int Scale::getScreenWidth() const {
	return screenWidth;
}

int Scale::getScreenHeight() const {
	return screenHeight;
}

__device__ float Scale::getWidthD() const {
	return width;
}

__device__ float Scale::getHeightD() const {
	return height;
}

__device__ int Scale::getScreenWidthD() const {
	return screenWidth;
}

__device__ int Scale::getScreenHeightD() const {
	return screenHeight;
}
