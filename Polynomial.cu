#include "hip/hip_runtime.h"
//
// Created by BabdCatha on 19/10/2021.
//

#include <iostream>
#include "Polynomial.cuh"

__device__
hipFloatComplex expRap(hipFloatComplex x, int i){
	if(i==0){
		return make_hipFloatComplex(1, 0);
	}else if(i == 1){
		return x;
	}else if(i%2 == 0){
		hipFloatComplex temp = expRap(x, i/2);
		return hipCmulf(temp, temp);
	}else{
		hipFloatComplex temp = expRap(x, i/2);
		return hipCmulf(x, hipCmulf(temp, temp));
	}
}

Polynomial::Polynomial(int degree, Root * rootsList) {
	this->degree = degree;
	roots = new Root[degree];
	for(int i = 0; i < degree; i++){
		roots[i] = rootsList[i];
	}
	//coefficients = new hipDoubleComplex[degree+1];
	hipMallocManaged(&coefficients, (degree+1)*sizeof(hipFloatComplex));
	computeCoefficients();
}

__device__
void Polynomial::evaluate(hipFloatComplex x, hipFloatComplex *res) {
	*res = make_hipFloatComplex(0, 0);
	for(int i = 0; i < degree+1; i++){
		//res += coefficients[i] * expRap(x, i);
		*res = hipCaddf(*res, hipCmulf(coefficients[i], expRap(x, i)));
	}
}

__device__
void Polynomial::evaluate_derivative(hipFloatComplex x, hipFloatComplex *res) {
	*res = make_hipFloatComplex(0, 0);
	for(int i = 1; i < degree+1; i++){
		//res += ((double)i*coefficients[i] * std::pow(x, i-1));
		*res = hipCaddf(*res, hipCmulf(make_hipFloatComplex(i, 0), hipCmulf(coefficients[i], expRap(x, i-1))));
	}
}

void Polynomial::computeCoefficients(){

	for(int i = 0; i < degree+1; i++){
		coefficients[i] = make_hipFloatComplex(0, 0);
	}

	coefficients[1] = make_hipFloatComplex(1, 0);
	//coefficients[0] = (double)-1*roots[0].getValue();
	coefficients[0] = hipCmulf(make_hipFloatComplex(-1, 0), roots[0].getValue());

	hipFloatComplex tempList[degree+1];

	for(int i = 1; i < degree; i++){

		for(int j = 1; j < degree+1 ; j++){
			tempList[j] = coefficients[j-1];
		}
		tempList[0] = make_hipFloatComplex(0, 0);

		for(int j = 0; j < degree+1; j++){
			//coefficients[j] *= (double)-1*roots[i].getValue();
			//coefficients[j] += tempList[j];

			coefficients[j] = hipCmulf(coefficients[j],hipCmulf(make_hipFloatComplex(-1, 0), roots[i].getValue()));
			coefficients[j] = hipCaddf(coefficients[j],tempList[j]);
		}
	}
}

bool Polynomial::getIsLeftMouseButtonPressed() const {
	return isLeftMouseButtonPressed;
}

//They first return a boolean indicating whether they are the one that was pressed. This is used to make
//sure that when the mouse is over several roots, only one is moved at any time.
void Polynomial::leftMouseButtonPressed(sf::Event event) {
	isLeftMouseButtonPressed = true;
	for(int i = 0; i < degree; i++){
		if(roots[i].overlaps(event.mouseButton.x, event.mouseButton.y)){
			roots[i].setSelected(true);
			break;
		}
	}
	//Debug code
	for(int i = 0; i < degree+1; i++){
		std::cout << i << " : " << hipCrealf(coefficients[i]) << "+i" << hipCimagf(coefficients[i]) << std::endl;
	}
}

void Polynomial::leftMouseButtonReleased() {
	isLeftMouseButtonPressed = false;
	//We release every root
	for(int i = 0; i < degree; i++){
		if(roots[i].isSelected()){
			roots[i].setSelected(false);
		}
	}
}

void Polynomial::update(sf::Event event) {
	for(int i = 0; i < degree; i++){
		if(roots[i].isSelected()){
			roots[i].updatePosition(event.mouseMove.x, event.mouseMove.y);
		}
	}
	computeCoefficients();
}

void Polynomial::drawRoots() {
	for(int i = 0; i < degree; i++){
		roots[i].draw();
	}
}

sf::Color Polynomial::findClosestRootColor(hipFloatComplex z){
	double dist = HUGE_VAL;
	sf::Color res;
	for(int i = 0; i < degree; i++){
		//double new_dist = std::abs(z - roots[i].getValue());
		double new_dist = hipCabsf(hipCsubf(z, roots[i].getValue()));
		if(new_dist < dist){
			dist = new_dist;
			res = roots[i].getRootColor();
		}
	}

	//Variable to be adjusted to make sure that the value is actually close to the root
	if(dist > 100.0){
		res = sf::Color::Black;
	}

	return res;

}